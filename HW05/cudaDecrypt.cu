#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


__device__ unsigned int modProdCuda(unsinged int a, unsigned int b, unsigned int p){
 
  unsigned int za = a;
  unsigned int ab = 0;

  while(b > 0){
    if(b%2 == 1) ab = (ab + za) % p;
    za =  (2 * za) % p;
    b /= 2;
  }
  return ab;
}

__device__ unsigned int modExpCuda(unsigned int a, unsigned int b, unsigned int p){
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
   if( (b%2 == 1) aExpb = modProdCuda(aExpb, z, p);
   z = modProdCuda(z, z, p);
   b /= 2;
  }
 
  return aExpb;
}


__global__ void kenrelFindSecretKey(unsigned int p, unsigned int g, unsigned int h, unsigned int *d_array){

  unsigned int thread = threadIdx.x; 
  unsigned int block = blockIdx.x;
  unsigned int blockSize = blockDim.x;
  //unsinged int gridsize =gridDim.x;

  //unique global thread id
  unsigned int id = thread + blockSize*block;  

  //for(unsinged int i=0; i<p-1; i++)
  if (id < p-1){
    if (modExpCuda(g, id+1, p) ==h) {
      d_array[0] = id+1;
    }
  }

}


int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  unsigned int n, p, g, h, x;
  unsigned int Nints;

  unsigned int *data_fp = (unsigned int *) malloc(4*sizeof(unsigned int));

  FILE *fp;
  FILE *fm;
 
  fp = fopen("public_key.txt", "r");
  fm = fopen("message.txt" , "r");

  for(unsigned int i = 0; i < 4; i++) {
    fscanf(fp, "%u", data_fp+i);
  }  

  fclose(fp);

  n = data_fp[0];
  p = data_fp[1];
  g = data_fp[2];
  h = data_fp[3];

  //secret key
  
  fscanf(fm, "%u", Nints);

  unsigned int *m_data = (int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a_data = (int *) malloc(Nints*sizeof(unsigned int));

  unsigned int *data_fm = (unsigned char*) malloc(Nints*sizeof(unsigned char));

  for(unsigned int j =0; j < Nints; j++){
    fscanf(fm, "%u %u", m_data+j, a_data+j);
}

  fclose(fm);

  //Cuda work goes here
  //unsigned int Nthreads
  //dim3 I();
  //hipMemcpy


  int bufferSize =1024;
  
  unsigned char *message = (unsigned char*) malloc(bufferSize*sizeof(unsigned char));
 
  unsigned int Nchars = ((n-1)/8)*Nints;

  ElGamaldecrypt(m_data, a_data, Nints, p , x);
  convertZtoString(m_data, Nints, message, Nchars);
  printf("The decrypted message: \"%s\" \n", message);

   return 0;

}
